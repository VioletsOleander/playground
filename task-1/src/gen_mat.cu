#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>


void GenMatFP16(int m, int n, __half* mat){
    srand(time(NULL));
    int cnt;
    float tmp;
    for(cnt=0; cnt<m*n; cnt++){
        tmp = (float)rand() / RAND_MAX;
        mat[cnt] = __half2float(tmp);
    }
}

void GenMatFP32(int m, int n, float *mat){
    srand(time(NULL));
    int cnt;
    for(cnt=0; cnt<m*n; cnt++){
        mat[cnt] = (float)rand() / RAND_MAX;
    }
}
