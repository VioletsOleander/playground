#include "hip/hip_runtime.h"
#include "parameters.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <stdio.h>

#define FRAG_M 16
#define FRAG_N 16
#define FRAG_K 16
#define FLOAT(POINTER) (reinterpret_cast<float *>(POINTER))

using namespace nvcuda;

__global__ void MatMulFP16(const int m, const int n, const int k,
                           half *a, half *b, half *c) {

    // size of matrix a: m * k (elements)
    // size of matrix b: k * n (elements)
    // size of matrix c: m * n (elements)

    half *gm_a = a;
    half *gm_b = b;
    half *gm_c = c;

    extern __shared__ half sm[];

    int smYStride_a = BLOCK_TILE_Y + FRAG_M;
    int smXStride_b = BLOCK_TILE_X + FRAG_N;

    half *sm_tileA = sm;                              // tileA shape(smYStride_a, BLOCK_TILE_K), size of tileA: (smYStride_a) * BLOCK_TILE_K (elements)
    half *sm_tileB = sm + smYStride_a * BLOCK_TILE_K; // tileB shape(BLOCK_TILE_K, smXStride_b), size of tileB: BLOCK_TILE_K * (smXStride_b) (elements)

    // one block computes BLOCK_TILE_X elements in x dim
    int globalIdxBlk_x = BLOCK_TILE_X * blockIdx.x;
    // one block computes BLOCK_TILE_Y elements in y dim
    int globalIdxBlk_y = BLOCK_TILE_Y * blockIdx.y;

    // threadIdx in block from sequential view
    int seqThreadIdx_blk = threadIdx.y * blockDim.y + threadIdx.x;
    int seqThreadIdx_warp = seqThreadIdx_blk % 32;
    int warpIdx = seqThreadIdx_blk / 32;

    int warpIdx_y = warpIdx / (BLOCK_TILE_X / WARP_TILE_X);
    int warpIdx_x = warpIdx % (BLOCK_TILE_X / WARP_TILE_X);

    int warpOffset_y = warpIdx_y * WARP_TILE_Y;
    int warpOffset_x = warpIdx_x * WARP_TILE_X;

    wmma::fragment<wmma::accumulator, FRAG_M, FRAG_N, FRAG_K, half> frag_c[WARP_TILE_Y / FRAG_M][WARP_TILE_X / FRAG_N];
#pragma unroll
    for (int i_frag = 0; i_frag < WARP_TILE_Y / FRAG_M; i_frag++) {
#pragma unroll
        for (int j_frag = 0; j_frag < WARP_TILE_X / FRAG_N; j_frag++) {
            wmma::fill_fragment(frag_c[i_frag][j_frag], 0);
        }
    }

    for (int i = 0; i < k / BLOCK_TILE_K; i++) { // exact division assumed

        // each thread fetch 2 continuous elements, each warp fetch 64 continuous elements
        // from gm_a[globalIdxBlk_y + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2][i * BLOCK_TILE_K + warpIdx + i_batch * N_WARP_PER_BLOCK]
        // to sm_tileA[(seqThreadIdx_warp  + WARP_SIZE * i_warp) * 2][warpIdx + i_batch * N_WARP_PER_BLOCK],
        // i_warp in [0, BLOCK_TILE_Y / WARP_SIZE / 2]
#pragma unroll
        for (int i_batch = 0; i_batch < BLOCK_TILE_K / N_WARP_PER_BLOCK; i_batch++) {
#pragma unroll
            for (int i_warp = 0; i_warp < BLOCK_TILE_Y / WARP_SIZE / 2; i_warp++) {
                FLOAT(sm_tileA + (warpIdx + i_batch * N_WARP_PER_BLOCK) * smYStride_a + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2)
                [0] = FLOAT(gm_a + (i * BLOCK_TILE_K + warpIdx + i_batch * N_WARP_PER_BLOCK) * k + globalIdxBlk_y + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2)[0];
            }
        }

        // fetch gm_b[i * BLOCK_TILE_K + warpIdx][globalIdxBlk_x + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2]
        // to sm_tileB[warpIdx + i_batch * N_WARP_PER_BLOCK][(seqThreadIdx + WARP_SIZE * i_warp) * 2],
        // i_warp in [0, BLOCK_TILE_X / WARP_SIZE / 2]
#pragma unroll
        for (int i_batch = 0; i_batch < BLOCK_TILE_K / N_WARP_PER_BLOCK; i_batch++) {
#pragma unroll
            for (int i_warp = 0; i_warp < BLOCK_TILE_X / WARP_SIZE / 2; i_warp++) {
                FLOAT(sm_tileB + (warpIdx + i_batch * N_WARP_PER_BLOCK) * smXStride_b + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2)
                [0] = FLOAT(gm_b + (i * BLOCK_TILE_K + warpIdx + i_batch * N_WARP_PER_BLOCK) * n + globalIdxBlk_x + (seqThreadIdx_warp + WARP_SIZE * i_warp) * 2)[0];
            }
        }

        __syncthreads(); // sync to make sure every element in sm_tileA and sm_tileB gets ready

        wmma::fragment<wmma::matrix_a, FRAG_M, FRAG_N, FRAG_K, half, wmma::col_major> frag_a[WARP_TILE_Y / FRAG_M];
        wmma::fragment<wmma::matrix_b, FRAG_M, FRAG_N, FRAG_K, half, wmma::row_major> frag_b[WARP_TILE_X / FRAG_N];

#pragma unroll
        for (int k_frag = 0; k_frag < BLOCK_TILE_K / FRAG_K; k_frag++) {
#pragma unroll
            for (int i_frag = 0; i_frag < WARP_TILE_Y / FRAG_M; i_frag++) {
                // sm_tileA[warpOffset_y + i_frag * FRAG_M (+FRAG_M)][k_frag * FRAG_K (+FRAG_N)]
                // to frag_a[i_frag]
                wmma::load_matrix_sync(frag_a[i_frag], sm_tileA + (k_frag * FRAG_K) * smYStride_a + warpOffset_y + i_frag * FRAG_M,
                                       smYStride_a);
            }
#pragma unroll
            for (int j_frag = 0; j_frag < WARP_TILE_X / FRAG_N; j_frag++) {
                // sm_tileB[k_frag * FRAG_K (+FRAG_M)][warpOffset_x + j_frag * FRAG_N (+FRAG_N)]
                wmma::load_matrix_sync(frag_b[j_frag], sm_tileB + (k_frag * FRAG_K) * smXStride_b + warpOffset_x + j_frag * FRAG_N,
                                       smXStride_b);
            }

#pragma unroll
            for (int i_frag = 0; i_frag < WARP_TILE_Y / FRAG_M; i_frag++) {
#pragma unroll
                for (int j_frag = 0; j_frag < WARP_TILE_X / FRAG_N; j_frag++) {
                    wmma::mma_sync(frag_c[i_frag][j_frag], frag_a[i_frag], frag_b[j_frag], frag_c[i_frag][j_frag]);
                }
            }
        }

        __syncthreads(); // sync to make sure every thread finished computation so sm_tileA and sm_tileB are allowed to be overwirtten
    }

#pragma unroll
    for (int i_frag = 0; i_frag < WARP_TILE_Y / FRAG_M; i_frag++) {
#pragma unroll
        for (int j_frag = 0; j_frag < WARP_TILE_X / FRAG_N; j_frag++) {
            // frag_c[i_frag][j_frag] to
            // gm_c[globalIdxBlk_y + warpOffset_y + i_frag * FRAG_M (+FRAG_M)][globalIdxBlk_x + warpOffset_x + j_frag * FRAG_N (+FRAG_N)]
            wmma::store_matrix_sync(gm_c + (globalIdxBlk_y + warpOffset_y + i_frag * FRAG_M) * n + globalIdxBlk_x + warpOffset_x + j_frag * FRAG_N,
                                    frag_c[i_frag][j_frag], n, wmma::mem_row_major);
        }
    }
}