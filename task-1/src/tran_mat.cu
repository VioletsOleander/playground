#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

void mat_32_16(int m, int n, float *a, __half *b){
    for(int i=0; i<m*n; i++){
        b[i] = __float2half(a[i]);
    }
}

void mat_16_32(int m, int n, __half *a, float *b){
    for(int i=0; i<m*n; i++){
        b[i] = __half2float(a[i]);
    }
}