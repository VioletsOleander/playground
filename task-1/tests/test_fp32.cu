#include "hip/hip_runtime.h"
#include "parameters.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void MatMulRef(const int, const int, const int, float *, int, float *, int, float *, int);
__global__ void MatMulFP32(const int, const int, const int, float *, float *, float *);
void GenMatFP32(int, int, float *);
float CompareMat(int, int, float *, float *);
void TransposeMatFP32(int, int, float *);

int main() {
    const int m = _M, n = _N, k = _K;
    const int lda = _K, ldb = _N, ldc = _N, ldr = _N;
    // printf("Matrix A: (%d * %d), Matrix B: (%d * %d), Matrix C: (%d * %d)\n",
    //    m, k, k, n, m, n);

    // allocate memory for matrices
    const size_t memSize_a = m * lda * sizeof(float);
    const size_t memSize_b = k * ldb * sizeof(float);
    const size_t memSize_c = m * ldc * sizeof(float);
    const size_t memSize_r = m * ldc * sizeof(float);
    float *h_a = (float *)malloc(memSize_a);
    float *h_b = (float *)malloc(memSize_b);
    float *h_c = (float *)malloc(memSize_c);
    float *h_r = (float *)malloc(memSize_r);

    // generate random matrices
    GenMatFP32(m, k, h_a);
    GenMatFP32(k, n, h_b);

    // get reference result
    MatMulRef(m, n, k, h_a, lda, h_b, ldb, h_r, ldr);

    TransposeMatFP32(n, k, h_a);
    // allocate memory in device
    float *d_a, *d_b, *d_r;

    hipMalloc((void **)&d_a, memSize_a);
    hipMalloc((void **)&d_b, memSize_b);
    hipMalloc((void **)&d_r, memSize_r);

    hipMemcpy(d_a, h_a, memSize_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memSize_b, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float runTime = 0.0, runTimeSum = 0.0;

    // configure kernel launch
    dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y);

    assert(BLOCK_DIM_X * THREAD_TILE_X == BLOCK_TILE_X);
    assert(BLOCK_DIM_Y * THREAD_TILE_Y == BLOCK_TILE_Y);
    const int numElementXDim_blk = BLOCK_TILE_X;
    const int numElementYDim_blk = BLOCK_TILE_Y;

    const int numElementXDim_grid = m;
    const int numElementYDim_grid = n;

    const int numBlockXDim_grid = (numElementXDim_grid + numElementXDim_blk - 1) / numElementXDim_blk;
    const int numBlockYDim_grid = (numElementYDim_grid + numElementYDim_blk - 1) / numElementYDim_blk;
    dim3 dimGrid(numBlockXDim_grid, numBlockYDim_grid);

    // const int offset_x = n / DIVIDER;
    // const int offset_y = m / DIVIDER;

    const int tileDim_m = numElementYDim_blk;
    const int tileDim_n = numElementXDim_blk;
    const int tileDim_k = BLOCK_TILE_K;

    // shared memory usage by each block tile
    size_t sMemPerBlk = (tileDim_m * tileDim_k + tileDim_n * tileDim_k) * N_PIPELINE_STAGE * sizeof(float);
    assert(sMemPerBlk < SM_PER_BLOCK);

    // run (N_REP+N_WARMUP) times
    for (int i = 0; i < (N_REP + N_WARMUP); i++) {
        // warm up
        if (i < N_WARMUP) {
            MatMulFP32<<<dimGrid, dimBlock, sMemPerBlk>>>(m, n, k, d_a, d_b, d_r);
            continue;
        }
        // run and timing N_REP times
        hipEventRecord(start, NULL);

        MatMulFP32<<<dimGrid, dimBlock, sMemPerBlk>>>(m, n, k, d_a, d_b, d_r);

        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&runTime, start, stop);
        runTimeSum += runTime;
    }

    hipMemcpy(h_c, d_r, memSize_r, hipMemcpyDeviceToHost);

    // compare result against reference
    float error = 0.0;
    error = CompareMat(m, n, h_r, h_c);

    // calculate tflops and average error
    float msecPerMatMul = runTimeSum / N_REP;
    double flopsPerMatMul = 2.0 * m * k * n;
    double tflops = (flopsPerMatMul * 1.0e-12f) / (msecPerMatMul / 1000.0f);

    printf("TFLOPS is: %lf\naverage error is: %f\n", tflops, error);

    // free device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_r);

    // free host memories
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_r);

    return 0;
}