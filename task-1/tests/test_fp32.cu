#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "parameters.h"

#include <hip/hip_runtime.h>

void MatMulRef(int, int, int, float *, int , float *, int, float *, int);
__global__ void MatMulFP32(int, int, int, float *, int, float *, int, float *, int);
void GenMatFP32(int, int, float*);
float CompareMat(int, int, float *, float *);

int main(){
    int m=M, n=N, k=K;
    int lda = k, ldb = n, ldr = n;
    float run_time = 0.0, sum_run_time = 0.0;
    float err = 0.0;
    float *a, *b, *r, *r_ref; 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate memory for matrices
    const size_t a_mem_size = m * k * sizeof(float);
    const size_t b_mem_size = k * n * sizeof(float);
    const size_t r_mem_size = m * n * sizeof(float);
    const size_t r_ref_mem_size = m * n * sizeof(float);
    a = (float*)malloc(a_mem_size);
    b = (float*)malloc(b_mem_size);
    r = (float*)malloc(r_mem_size);
    r_ref = (float*)malloc(r_ref_mem_size);
    //generate random matrices
    GenMatFP32(m, k, a);
    GenMatFP32(k, n, b);

    //get benchmark
    MatMulRef(m, n, k, a, lda, b, ldb, r_ref, ldr); 

    //allocate memory in device
    float *d_A, *d_B, *d_R;
    hipMalloc((void**)&d_A, a_mem_size);
    hipMalloc((void**)&d_B, b_mem_size);
    hipMalloc((void**)&d_R, r_mem_size);
    
    hipMemcpy(d_A, a, a_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, b_mem_size, hipMemcpyHostToDevice);

    //run for (N_REP+N_WARMUP) times
    for(int i=0; i<(N_REP+N_WARMUP); i++){
        //warm up
        if (i<N_WARMUP){
            MatMulFP32<<<1024, 64>>>(m, n, k, d_A, lda, d_B, ldb, d_R, ldr);
            continue;
        }
        //running and timing  N_REP times
        hipEventRecord(start, NULL);

        MatMulFP32<<<1024, 64>>>(m, n, k, d_A, lda, d_B, ldb, d_R, ldr);

        hipEventRecord(stop, NULL); 
        hipEventSynchronize(stop);
        hipEventElapsedTime(&run_time, start, stop);
        sum_run_time += run_time;
    }

    //compare result and benchmark
    hipMemcpy(r, d_R, r_mem_size, hipMemcpyDeviceToHost);
    err = CompareMat(m, n, r_ref, r);

     //calculate tflops and average error
    float msecPerMatrixMul = sum_run_time / N_REP;
    double flopsPerMatrixMul = 2.0 * m * k * n;
    double tflops = (flopsPerMatrixMul * 1.0e-12f) / (msecPerMatrixMul / 1000.0f);

    printf("TFLOPS is: %lf\naverage error is: %f\n", tflops, err);

    //free memories in device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_R);
        
    //free memories in host
    free(a);
    free(b);
    free(r);
    free(r_ref);

    return 0;
}