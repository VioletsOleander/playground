#include "hip/hip_runtime.h"
#include "parameters.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void MatMulRef(const int, const int, const int, float *, int, float *, int, float *, int);
__global__ void MatMulFP32(const int, const int, const int, const float *, const int, const float *, const int, float *, const int);
void GenMatFP32(int, int, float *);
float CompareMat(int, int, float *, float *);

int main() {
    const int m = M, n = N, k = K;
    const int lda = K, ldb = N, ldc = N, ldr = N;
    // printf("Matrix A: (%d * %d), Matrix B: (%d * %d), Matrix C: (%d * %d)\n",
    //    m, k, k, n, m, n);

    // allocate memory for matrices
    const size_t memSize_a = m * lda * sizeof(float);
    const size_t memSize_b = k * ldb * sizeof(float);
    const size_t memSize_c = m * ldc * sizeof(float);
    const size_t memSize_r = m * ldc * sizeof(float);
    float *h_a = (float *)malloc(memSize_a);
    float *h_b = (float *)malloc(memSize_b);
    float *h_c = (float *)malloc(memSize_c);
    float *h_r = (float *)malloc(memSize_r);

    // generate random matrices
    GenMatFP32(m, k, h_a);
    GenMatFP32(k, n, h_b);

    // get reference result
    MatMulRef(m, n, k, h_a, lda, h_b, ldb, h_r, ldr);

    // allocate memory in device
    float *d_a, *d_b, *d_r;

    hipMalloc((void **)&d_a, memSize_a);
    hipMalloc((void **)&d_b, memSize_b);
    hipMalloc((void **)&d_r, memSize_r);

    hipMemcpy(d_a, h_a, memSize_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memSize_b, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float runTime = 0.0, runTimeSum = 0.0;

    // configure kernel launch
    const int numWarp_blk = 8;
    const int numThread_blk = numWarp_blk * N_THR_PER_WARP;
    const int numThreadXDim_blk = sqrt((double)numThread_blk);
    const int numThreadYDim_blk = numThread_blk / numThreadXDim_blk; // exact division assumed
    dim3 dimBlock(numThreadXDim_blk, numThreadYDim_blk);

    const int numThreadXDim_grid = m; // each thread responsible for one output
    const int numThreadYDim_grid = n;
    const int numBlockXDim_grid = (numThreadXDim_grid + numThreadXDim_blk - 1) / numThreadXDim_blk;
    const int numBlockYDim_grid = (numThreadYDim_grid + numThreadYDim_blk - 1) / numThreadYDim_blk;
    dim3 dimGrid(numBlockXDim_grid, numBlockYDim_grid);
    // printf("Grid dim: (%d, %d), Block dim: (%d, %d)\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

    const int tileDim_m = numThreadYDim_blk;
    const int tileDim_n = numThreadXDim_blk;
    assert(tileDim_m == tileDim_n); // square tile block assumed
    const int tileDim_k = WIDTH_BLOCK_TILE;
    // printf("Blocktile A dim: (%d, %d), BlockTile B dim: (%d, %d), Blocktile C dim: (%d, %d)\n",
    //    tileDim_m, tileDim_k, tileDim_k, tileDim_n, tileDim_m, tileDim_n);
    // shared memory usage by each block tile
    size_t sMemPerBlk = (tileDim_m * tileDim_k + tileDim_n * tileDim_k) * sizeof(float);
    // printf("Shared memory usage: %d bytes per block\n", sMemPerBlk);
    assert(sMemPerBlk < SM_PER_BLOCK);

    // run (N_REP+N_WARMUP) times
    for (int i = 0; i < (N_REP + N_WARMUP); i++) {
        // warm up
        if (i < N_WARMUP) {
            MatMulFP32<<<dimGrid, dimBlock, sMemPerBlk>>>(m, n, k, d_a, lda, d_b, ldb, d_r, ldr);
            continue;
        }
        // run and timing N_REP times
        hipEventRecord(start, NULL);

        MatMulFP32<<<dimGrid, dimBlock, sMemPerBlk>>>(m, n, k, d_a, lda, d_b, ldb, d_r, ldr);

        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&runTime, start, stop);
        runTimeSum += runTime;
    }

    hipMemcpy(h_c, d_r, memSize_r, hipMemcpyDeviceToHost);

    // compare result against reference
    float error = 0.0;
    error = CompareMat(m, n, h_r, h_c);

    // calculate tflops and average error
    float msecPerMatMul = runTimeSum / N_REP;
    double flopsPerMatMul = 2.0 * m * k * n;
    double tflops = (flopsPerMatMul * 1.0e-12f) / (msecPerMatMul / 1000.0f);

    printf("TFLOPS is: %lf\naverage error is: %f\n", tflops, error);

    // free device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_r);

    // free host memories
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_r);

    return 0;
}